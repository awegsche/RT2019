#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "bvh.h"
#include "triangle.h"
#include "commonheader.h"
#include "../Utils/cuda_helpers.h" 

#include <thrust/device_vector.h>

//#if __CUDA_ARCH__ < 600
//__device__ double atomicAdd(double* address, double val)
//{
//    unsigned long long int* address_as_ull =
//                              (unsigned long long int*)address;
//    unsigned long long int old = *address_as_ull, assumed;
//
//    do {
//        assumed = old;
//        old = atomicCAS(address_as_ull, assumed,
//                        __double_as_longlong(val +
//                               __longlong_as_double(assumed)));
//
//    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
//    } while (assumed != old);
//
//    return __longlong_as_double(old);
//}
//#endif

namespace geometry {

	const int BlockSize = 256;

	__host__ __device__ AABB tri_bounding_box(const Triangle &tri, const vec3* vertices) {
		return AABB(
			min(min(vertices[tri.v1].x, vertices[tri.v2].x), vertices[tri.v3].x),
			min(min(vertices[tri.v1].y, vertices[tri.v2].y), vertices[tri.v3].y),
			min(min(vertices[tri.v1].z, vertices[tri.v2].z), vertices[tri.v3].z),

			max(max(vertices[tri.v1].x, vertices[tri.v2].x), vertices[tri.v3].x),
			max(max(vertices[tri.v1].y, vertices[tri.v2].y), vertices[tri.v3].y),
			max(max(vertices[tri.v1].z, vertices[tri.v2].z), vertices[tri.v3].z)
		);
	}

	AABB AABB::unite(const AABB & a, const AABB & b)
	{
		return AABB(
			vec3(min(a.p0.x, b.p0.x), min(a.p0.y, b.p0.y), min(a.p0.z, b.p0.z)),
			vec3(max(a.p1.x, b.p1.x), max(a.p1.y, b.p1.y), max(a.p1.z, b.p1.z))
		);
	}


	__global__ void construct_bvh(BVHNode *nodes, BVHNode* leaves,
		Triangle* tris, vec3* vertices, int numtriangles, int* nodeCounter)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;

		if (i < numtriangles) {
			BVHNode* leaf = leaves + i;

			// Handle leaf first
			leaf->minId = i;
			//printf("%d, %d\n", leaf->minId, (leaves + i)->minId);
			leaf->boundin_box = tri_bounding_box(tris[i], vertices);

			uint current = leaf->parent;

			int res = atomicAdd(nodeCounter + current, 1);
		printf("%d\n", i);

			// Go up and handle internal nodes
			while (true) {
				if (res == 0) {
					return;
				}
				BVHNode* currentNode = nodes + current;
				AABB leftBoundingBox = nodes[currentNode->left].boundin_box;
				AABB rightBoundingBox = nodes[currentNode->right].boundin_box;

				// Compute current bounding box
				currentNode->boundin_box = AABB::unite(leftBoundingBox,
											  rightBoundingBox);
				vec3 sides = currentNode->boundin_box.sides();
				printf("%f, %f, %f\n", sides.x, sides.y, sides.z);

				// If current is root, return
				if (current == 0) {
					return;
				}
				current = currentNode->parent;
				res = atomicAdd(nodeCounter + current, 1);
			}
		}
	}

	void cuda_construct_bvh(const BVH& bvh)
	{
		int blockSize = BlockSize;
		int gridSize = (bvh.numTriangles + blockSize - 1) / blockSize;
		thrust::device_vector<int> nodeCounter(bvh.numTriangles);
		construct_bvh << < 1, bvh.numTriangles >> > (bvh.device_nodes, bvh.device_leaves,
			bvh.device_tris, bvh.device_vertices, bvh.numTriangles, nodeCounter.data().get());

		cudaLAUNCH(hipDeviceSynchronize())
	}


}
